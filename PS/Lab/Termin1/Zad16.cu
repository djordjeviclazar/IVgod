#include "hip/hip_runtime.h"
%%cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>

#define N 20

__global__ void kernel1(int* A, int* B, int* C)
{
    int length = N * N;
    int index = threadIdx.x, pom;

    while(index < length)
    {
        pom = A[index] - B[index];
        C[index] = pom < 0 ? 0 : pom;
     
        index += blockDim.x;
    }
}

__global__ void kernel2(int* A, int* B, int* C)
{
    int length = N * N;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int threadNumber = blockDim.x * gridDim.x, pom;

    while(index < length)
    {
        pom = A[index] - B[index];
        C[index] = pom < 0 ? 0 : pom;
     
        index += threadNumber;
    }
}

bool test(int* A, int* B, int* C)
{
    int length = N * N;
 
    for(int i = 0; i < length; i++)
    {
       int result = A[i] - B[i];
       if (result < 0) { result = 0; }
       if (result != C[i])
       {
           return false;
       }
    }
 
    return true;
}

int* generateMatrix()
{
    const int length = N * N;
 
    int* result = new int[length];
    
    for(int i = 0; i < N * N; i++)
    {
        result[i] = rand() % 100;
    }
 
    return result;
}

void printMatrix(int* A)
{
    int length = N * N, k = 0;
    for(int i = 0; i < length; i++)
    {
        std::cout << A[i] << "\t";

        if(++k >= N)
        {
            k = 0;
            std::cout << "\r\n";
        }
        
    }
 
    std::cout << "-----------------------------------------\r\n";
}

int main(void)
{
    srand(time(0));
    const int length = N * N;
    int *A = generateMatrix(), *B = generateMatrix(), *C = new int[length]; 
    int *Adev, *Bdev, *Cdev ;
  	bool result = false;
 
    hipMalloc(&Adev, length * sizeof(int));
    hipMalloc(&Bdev, length * sizeof(int));
    hipMalloc(&Cdev, length * sizeof(int));
 
    hipMemcpy(Adev, A, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Bdev, B, length * sizeof(int), hipMemcpyHostToDevice);

    kernel1<<<1, 20>>>(Adev, Bdev, Cdev);
    hipMemcpy(C, Cdev, length * sizeof(int), hipMemcpyDeviceToHost);
 
    result = test(A, B, C);
    if(result) { std::cout << "Correct"; }
    else { std::cout << "Incorrect"; }
 
    std::cout << "\r\n";
    printMatrix(A);
    std::cout << "\r\n";
    printMatrix(B);
    std::cout << "\r\n";
    printMatrix(C);
	
    kernel2<<<20, 1>>>(Adev, Bdev, Cdev);
    hipMemcpy((void*)C, (void*)Cdev, length * sizeof(int), hipMemcpyDeviceToHost);
 
    result = test(A, B, C);
    if(result) { std::cout << "Correct"; }
    else { std::cout << "Incorrect"; }
    std::cout << "\r\n";
 
    kernel2<<<5, 4>>>(Adev, Bdev, Cdev);
    hipMemcpy((void*)C, (void*)Cdev, length * sizeof(int), hipMemcpyDeviceToHost);
 
    result = test(A, B, C);
    if(result) { std::cout << "Correct"; }
    else { std::cout << "Incorrect"; }
    std::cout << "\r\n";
 
    kernel2<<<2, 10>>>(Adev, Bdev, Cdev);
    hipMemcpy((void*)C, (void*)Cdev, length * sizeof(int), hipMemcpyDeviceToHost);
 
    result = test(A, B, C);
    if(result) { std::cout << "Correct"; }
    else { std::cout << "Incorrect"; }
    std::cout << "\r\n";
 
    kernel2<<<10, 2>>>(Adev, Bdev, Cdev);
    hipMemcpy((void*)C, (void*)Cdev, length * sizeof(int), hipMemcpyDeviceToHost);

    result = test(A, B, C);
    if(result) { std::cout << "Correct"; }
    else { std::cout << "Incorrect"; }
    std::cout << "\r\n";
 
    /*free*/
    delete[] A; delete[] B; delete[] C;
    A = 0; B = 0; C = 0;

    hipFree(Adev);
    hipFree(Bdev);
    hipFree(Cdev);

    hipDeviceReset();

    return 0;
}